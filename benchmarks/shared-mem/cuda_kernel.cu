#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include "interfaces/cuda/Internals.h"

#define WARP_SIZE 32


// https://docs.nvidia.com/cuda/inline-ptx-assembly/index.html
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
__device__ void copy(float *in, float *out) {
  uint64_t tid = threadIdx.x;

  #pragma unroll
  for (size_t r{0}; r < REPEATS; ++r) {
    asm volatile ("{\t\n"
                  ".reg .f32 data;\n\t"
                  ".reg .u64 r0;\n\t"
                  ".reg .u64 r1;\n\t"
                  "cvta.to.shared.u64 r0, %0;\n\t"
                  "cvta.to.shared.u64 r1, %1;\n\t"
                  "ld.shared.f32 data, [r1];\n\t"
                  "st.shared.f32 [r0], data;\n\t"
                  "}" :: "l"(out + tid), "l"(in + tid) : "memory");

  }
}


__global__ void kernel_shrMemBW(uint32_t *clocks, float* scratch, float magicNumber) {
  __shared__ float in[WARP_SIZE];
  __shared__ float out[WARP_SIZE];

  uint64_t tid = threadIdx.x;
  in[tid] = magicNumber;
  asm volatile ("bar.warp.sync  0xffffffff;\n\t" ::);

  // warm up
  copy(in, out);
  asm volatile ("bar.warp.sync  0xffffffff;\n\t" ::);


  // measure
  uint32_t start{0};
  uint32_t end{0};
  asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
  copy(in, out);
  asm volatile ("bar.warp.sync  0xffffffff;\n\t" ::);
  asm volatile ("mov.u32 %0, %%clock;" : "=r"(end) :: "memory");

  scratch[tid] = out[tid];
  if (tid == 0) {
    clocks[START] = start;
    clocks[END] = end;
  }
}


void shrMemBW(uint32_t *clocks, float* scratch, float magicNumber) {
  dim3 block(WARP_SIZE, 1, 1);
  dim3 grid(1, 1, 1);
  kernel_shrMemBW<<<grid, block>>>(clocks, scratch, magicNumber); CHECK_ERR;

}


double getPeakGPUFrequency(int deviceId) {
  int peakClock{0};
  hipDeviceGetAttribute(&peakClock, hipDeviceAttributeClockRate, deviceId); CHECK_ERR;
  // Note: cure returns frequency in kHz
  return static_cast<double>(peakClock * 1000);
}


long long getNumTransferedBytes() {
  constexpr long long numReadWritePerInteration{2};
  return static_cast<long long>(numReadWritePerInteration * WARP_SIZE * REPEATS * sizeof(float));
}


size_t getLaneSize() {
  return WARP_SIZE;
};

