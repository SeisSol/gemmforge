
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>

#define CHECK_ERR checkErr(__FILE__,__LINE__)

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

std::string PrevFile = "";
int PrevLine = 0;

void checkErr(const std::string &File, int Line) {
#ifndef NDEBUG
      hipError_t Error = hipGetLastError();
      if (Error != hipSuccess) {
        std::cout << std::endl << File
                  << ", line " << Line
                  << ": " << hipGetErrorString(Error)
                  << " (" << Error << ")"
                  << std::endl;

        if (PrevLine > 0)
          std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
        throw;
      }
      PrevFile = File;
      PrevLine = Line;
#endif
}

// Dense x Dense Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 81 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[612];
      float * localShrMem0 = &totalShrMem[612 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedTransposePatchLoader
      {
        int index;
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
          index = threadIdx.x + i * 64;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + i * 64];
        }
        if (threadIdx.x < 19) {
          index = threadIdx.x + 512;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + 512];
        }
      }

      float* shrRegion1 = &localShrMem0[531];
      // using ExtendedPatchLoader
      {
        shrRegion1[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        if (threadIdx.x < 17) {
          shrRegion1[threadIdx.x + 64] = glb_B[threadIdx.x + 64];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        for (int k = 0; k < 9; ++k) {
          value = shrRegion0[threadIdx.x + k * 59];

          #pragma unroll
          for (int n = 0; n < 9; ++n) {
            reg0[n] += value * shrRegion1[k + 9 * n];
          }
        }
      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Sparse Kernel
__global__ void 
__launch_bounds__(64)
 kernel_sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_b15c35b(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags) {
  unsigned batchID = (threadIdx.y + blockDim.y * blockIdx.x);
  if (batchID < numElements) {
    bool isFlagsProvided = (flags != nullptr);
    bool allowed = isFlagsProvided ? static_cast<bool>(flags[batchID]) : true;
    if (allowed) {
      const float * const __restrict__ glb_A = &A[batchID * 504 + 0 + A_extraOffset];
      const float * const __restrict__ glb_B = &B[batchID * 9 + 0 + B_extraOffset];
      float * const __restrict__ glb_C = &C[batchID * 504 + 0 + C_extraOffset];
      float reg0[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
      __shared__  __align__(8) float totalShrMem[540];
      float * localShrMem0 = &totalShrMem[540 * threadIdx.y];

      float* shrRegion0 = &localShrMem0[0];
      // using ExtendedTransposePatchLoader
      {
        int index;
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
          index = threadIdx.x + i * 64;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + i * 64];
        }
        if (threadIdx.x < 19) {
          index = threadIdx.x + 512;
          shrRegion0[(index % 9) * 59 + index / 9] = glb_A[threadIdx.x + 512];
        }
      }

      float* shrRegion1 = &localShrMem0[531];
      // using ExtendedPatchLoader
      {
        if (threadIdx.x < 9) {
          shrRegion1[threadIdx.x + 0] = glb_B[threadIdx.x + 0];
        }
      }
      __syncthreads();
      if (threadIdx.x < 56) {
        float value;

        value = shrRegion0[threadIdx.x + 0 * 59];

        // Mul begin col 0
        reg0[1] += value * shrRegion1[0];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 1 * 59];

        // Mul begin col 1
        reg0[1] += value * shrRegion1[1];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 2 * 59];

        // Mul begin col 2
        reg0[1] += value * shrRegion1[2];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 3 * 59];

        // Mul begin col 3
        reg0[1] += value * shrRegion1[3];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 4 * 59];

        // Mul begin col 4
        reg0[1] += value * shrRegion1[4];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 5 * 59];

        // Mul begin col 5
        reg0[1] += value * shrRegion1[5];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 6 * 59];

        // Mul begin col 6
        reg0[1] += value * shrRegion1[6];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 7 * 59];

        // Mul begin col 7
        reg0[1] += value * shrRegion1[7];
        // Mul end col 1

        value = shrRegion0[threadIdx.x + 8 * 59];

        // Mul begin col 8
        reg0[1] += value * shrRegion1[8];
        // Mul end col 1

      }
      if (threadIdx.x < 56) {
        #pragma unroll
        for (int n = 0; n < 9; ++n) {
          glb_C[threadIdx.x + 56 * n] = reg0[n] + glb_C[threadIdx.x + 56 * n];
        }
      }
    }
  }
}


// Dense x Dense Kernel Launcher
void sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}


// Dense x Sparse Kernel Launcher
void sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_b15c35b(const float * A, int A_extraOffset, const float * B, int B_extraOffset, float * C, int C_extraOffset, unsigned numElements, unsigned* flags, void* streamPtr) {
  dim3 block(64, 1, 1);
  dim3 grid((numElements + 1 - 1) / 1, 1, 1);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  kernel_sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_b15c35b<<<grid,block,0,stream>>>(A, A_extraOffset, B, B_extraOffset, C, C_extraOffset, numElements, flags);
  CHECK_ERR;
}



int main(){
  float A[9*56] = {1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.,
 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1., 1.};
  float B_sparse[9] = {4., 4., 4., 4., 4., 4., 4., 4., 4.};
  float B_dense[9 * 9] = {0., 0., 0., 0., 0., 0., 0., 0., 0., 4., 4., 4., 4., 4., 4., 4., 4., 4.,
 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0.,
 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0.,
 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0., 0.,
 0., 0., 0., 0., 0., 0., 0., 0., 0.};
  float C[56*9] = {0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1,
 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1, 0.1};
  float R1[56*9];
  float R2[56*9];

  float *A_dev = nullptr;
  float *B_sparse_dev = nullptr;
  float *B_dense_dev = nullptr;
  float *C1_dev = nullptr;
  float *C2_dev = nullptr;

  hipMalloc((void **)&A_dev, sizeof(float) * 9 * 56); CHECK_ERR;
  hipMalloc((void **)&B_sparse_dev, sizeof(float) * 9); CHECK_ERR;
  hipMalloc((void **)&B_dense_dev, sizeof(float) * 9 * 9); CHECK_ERR;
  hipMalloc((void **)&C1_dev, sizeof(float) * 56 * 9); CHECK_ERR;
  hipMalloc((void **)&C2_dev, sizeof(float) * 56 * 9); CHECK_ERR;

  hipMemcpy((void *)A_dev, (void *)A, sizeof(float) * 9 * 56, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_sparse_dev, (void *)B_sparse, sizeof(float) *  9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)B_dense_dev, (void *)B_dense, sizeof(float) *  9 * 9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C1_dev, (void *)C, sizeof(float) * 56 * 9, hipMemcpyHostToDevice); CHECK_ERR;
  hipMemcpy((void *)C2_dev, (void *)C, sizeof(float) * 56 * 9, hipMemcpyHostToDevice); CHECK_ERR;

  // Dense x Dense Matrix Mult
   sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_0b51dc0(A_dev, 0, B_dense_dev, 0, C1_dev, 0, 1, nullptr, nullptr);
  hipDeviceSynchronize();
  hipMemcpy(R1, C1_dev, sizeof(float)*56*9, hipMemcpyDeviceToHost);

  // Dense x Sparse Matrix Mult
   sgemm_T_NT_m9_n9_k56_lda9_ldb9_ldc56_alpha_1_beta_1_sss_b15c35b(A_dev, 0, B_sparse_dev, 0, C2_dev, 0, 1, nullptr, nullptr);
  hipDeviceSynchronize();
  hipMemcpy(R2, C2_dev, sizeof(float)*56*9, hipMemcpyDeviceToHost);

  std::cout << "[";
  for (int ii = 0; ii < 56*9 -1; ii++){
    std::cout << R1[ii] << ", ";
  }
  std::cout << R1[56*9 -1] << "]" << std::endl;
  std::cout << "[";
  for (int ii = 0; ii < 56*9 - 1; ii++){
    std::cout << R2[ii] << ", ";
  }
  std::cout << R2[56*9 -1] << "]" << std::endl;
  for (int i = 0; i < 56*9; i++){
    if (R1[i] != R2[i]) {
    throw std::runtime_error("Transposed Dense x  Dense and Transposed Dense x  Sparse Matrix Mismatch in Multiplication!");
    }
  }
  std::cout << "Transposed Dense x  Dense and Transposed Dense x  Sparse Matrix Multiplications Match!" << std::endl;
}
